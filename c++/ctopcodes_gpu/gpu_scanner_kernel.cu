
#include <hip/hip_runtime.h>
#define DIVUP(a,b) (((a%b)==0)?(a/b):((a/b)+1))
#define SECTORS 13
#define PI      3.14159265f
#define ARC     ((2.0f*PI)/(float)SECTORS)
#define WIDTH   8
#define ARCS    10 
#define COLS_PER_THREAD 16 // 16 gives performance of 45ms
#define BLOCKWIDTH 16
#define BLOCKHEIGHT 16

__device__ inline float _fround(float v) {
	return (float)((int)((v < 0.0f) ? v - 0.5f : v + 0.5f));
}
__device__ inline int _iround(float v) {
	return (int)((v < 0.0f) ? v - 0.5f : v + 0.5f);
}
__device__ unsigned int getBW3x3(const unsigned int *data, int x, int y, int width, int height) {
	if (x < 1 || x > width-2 || y < 1 || y >= height-2) return 0;
	unsigned int pixel, sum = 0;    
	#pragma unroll
	for (int j=y-1; j<=y+1; j++) {
		#pragma unroll
		for (int i=x-1; i<=x+1; i++) {
			pixel = data[j * width + i];
			sum += ((pixel >> 24) & 0x01);
		}
	}
	return (sum >= 5) ? 1 : 0;
}
__device__ int xdist(const unsigned int *data, int x, int y, int d, int width, int height) {
	unsigned int sample;
	unsigned int start = getBW3x3(data, x, y, width, height);
	#pragma unroll
	for (int i=x+d; i>1 && i<width-1; i+=d) {
		sample = getBW3x3(data, i, y, width, height);
		if (start + sample == 1) { 
			return (d > 0) ? i - x : x - i;
		}
	}
	return -1;
}
__device__ int ydist(const unsigned int *data, int x, int y, int d, int width, int height) {
	unsigned int sample;
	unsigned int start = getBW3x3(data, x, y, width, height);
	#pragma unroll
	for (int j=y+d; j>1 && j<height-1; j+=d) {
		sample = getBW3x3(data, x, j, width, height);
		if (start + sample == 1) {			
			return (d > 0) ? j - y : y - j;
		}
	}
	return -1;
}

__device__ float readUnit(const unsigned int *data, float x, float y, int width, int height) {
	int sx = _iround(x);
	int sy = _iround(y);
	bool whiteL = true;
	bool whiteR = true;
	bool whiteU = true;
	bool whiteD = true;  
	unsigned int sample;
	int distL = 0, distR = 0, distU = 0, distD = 0;
	#pragma unroll
	for (int i=1; true; i++) {
		if (sx - i < 1 || sx + i >= width - 1 ||
			sy - i < 1 || sy + i >= height - 1 ||
			i > 100) {
			return -1;
		}
		// Left sample
		sample = getBW3x3(data, sx - i, sy, width, height);
		if (distL <= 0) { 
			if (whiteL && sample == 0) {
				whiteL = false;
			} else if (!whiteL && sample == 1) {
				distL = i;
			}
		}
		// Right sample
		sample = getBW3x3(data, sx + i, sy, width, height);
		if (distR <= 0) { 
			if (whiteR && sample == 0) {
				whiteR = false;
			} else if (!whiteR && sample == 1) {
				distR = i;
			}
		}

		// Up sample
		sample = getBW3x3(data, sx, sy - i, width, height);
		if (distU <= 0) {
			if (whiteU && sample == 0) {
				whiteU = false;
			} else if (!whiteU && sample == 1) {
				distU = i;
			}
		}
	 
		// Down sample
		sample = getBW3x3(data, sx, sy + i, width, height);
		if (distD <= 0) {
			if (whiteD && sample == 0) {
				whiteD = false;
			} else if (!whiteD && sample == 1) {
				distD = i;
			}
		}

		if (distR > 0 && distL > 0 && distU > 0 && distD > 0) {
			return (float)(distR + distL + distU + distD) / 8.0f;
		}
	}
}
__device__ unsigned int getSample3x3(const unsigned int *data, int x, int y, int w, int h) {
	if (x < 1 || x > w-2 || y < 1 || y >= h-2) return 0;
	unsigned int sum = 0;      
	#pragma unroll
	for (int j=y-1; j<=y+1; j++) {
		#pragma unroll
		for (int i=x-1; i<=x+1; i++) {
			if ((data[j * w + i] & (unsigned int)0x01000000) > 0) {
				sum += (unsigned int)0xff;
			}
		}
	}
	return (sum / 9);
}
__device__ bool checksum(unsigned int bits) {
	unsigned int sum = 0;
	#pragma unroll
	for (int i=0; i<SECTORS; i++) {
		sum += (bits & (unsigned int)0x01);
		bits = bits >> 1;
	}
	return (sum == (unsigned int)5);
}
/** return uint2(c, code) */
__device__ uint2 readCode(const unsigned int *data, float unit, float arca, float x, float y, int w, int h) {
	float dx, dy;  // direction vector
	float dist;
	int c = 0;
	int sx, sy;
	int bit;
	unsigned bits = 0;
	int code = -1;
	unsigned int core[WIDTH];
	#pragma unroll
	for (int sector = SECTORS-1; sector >= 0; sector--) {
		dx = (float)cos(ARC * (float)sector + arca);
		dy = (float)sin(ARC * (float)sector + arca);
		// Take 8 samples across the diameter of the symbol
		#pragma unroll
		for (int i=0; i<WIDTH; i++) {
			dist = ((float)i - 3.5f) * unit;
			sx = (int)_fround(x + dx * dist);
			sy = (int)_fround(y + dy * dist);
			core[i] = getSample3x3(data, sx, sy, w, h);
		}

		// white rings
		if (core[1] <= 128 || core[3] <= 128 ||	core[4] <= 128 || core[6] <= 128) {
			return make_uint2(0,0);
		}

		// black ring
		if (core[2] > 128 || core[5] > 128) {
			return make_uint2(0,0);
		}

		// compute confidence in core sample
		c += (core[1] + core[3] + core[4] + core[6] + // white rings
		(0xff - core[2]) + (0xff - core[5]));  // black ring

		// data rings
		c += abs((int)core[7] * 2 - 0xff);

		// opposite data ring
		c += (0xff - abs((int)core[0] * 2 - 0xff));

		bit = (core[7] > 128)? 1 : 0;
		bits <<= 1;
		bits += bit;
	}
	if (checksum(bits)) {
		code = bits;
		return make_uint2(c, code);
	} else {
		return make_uint2(0,0);
	}
}
__device__ unsigned int rotateLowest(unsigned int bits, float arca) {
	unsigned int _min = bits;
	int mask = 0x1fff;
	// slightly overcorrect arc-adjustment ideal correction would be (ARC / 2),
	// but there seems to be a positive bias that falls out of the algorithm.
	arca -= (ARC * 0.65f);      
	float orientation = 0;
	#pragma unroll
	for (int i=1; i<=SECTORS; i++) {
		bits = (((bits << 1) & mask) | (bits >> (SECTORS - 1)));
		if (bits < _min) { 
			_min = bits;
			orientation = ((float)i * -ARC);
		}
	}
	orientation += arca;
	return _min;
}   
__device__ unsigned int cuda_krnl_decode(const unsigned int *data, int cx, int cy, int width, int height) {
	int up =	ydist(data, cx    , cy    , -1, width, height) + 
				ydist(data, cx - 1, cy    , -1, width, height) + 
				ydist(data, cx + 1, cy    , -1, width, height);
	int down =  ydist(data, cx    , cy    , 1 , width, height) +
				ydist(data, cx - 1, cy    , 1 , width, height) +
				ydist(data, cx + 1, cy    , 1 , width, height);
	int left =  xdist(data, cx    , cy    , -1, width, height) +
				xdist(data, cx    , cy - 1, -1, width, height) +
				xdist(data, cx    , cy + 1, -1, width, height);
	int right = xdist(data, cx    , cy    , 1 , width, height) +
				xdist(data, cx    , cy - 1, 1 , width, height) +
				xdist(data, cx    , cy + 1, 1 , width, height);
	float x = (float)cx;
	float y = (float)cy;
	x += (right - left) / 6.0f;
	y += (down - up) / 6.0f;
	int unit = readUnit(data, x, y, width, height);
	if (unit < 0) 
		return -1;

	unsigned int code = 0;
	unsigned int c = 0;
	int   maxc = 0;
	float arca;
	float maxa = 0;
	float maxu = 0;
	
	//-----------------------------------------
	// Try different unit and arc adjustments,
	// save the one that produces a maximum
	// confidence reading...
	//-----------------------------------------
	float dARC = 1.0f/(float)ARCS;
	float hdARC = dARC/2.0f;
	uint2 ui2;
    #pragma unroll
	for (int u = -2; u <= 2; u++) { 
		#pragma unroll
		for (int a = 0; a < ARCS; a++) {
			arca = (float)a * ARC * dARC;
			ui2 = readCode(data, unit + (unit * hdARC * u), arca, x, y, width, height);			
			c = ui2.x;
			code = ui2.y;
			if (c > maxc) { 
				maxc = c;
				maxa = arca;
				maxu = unit + (unit * hdARC * u);
			}
		}
	}
	
	// One last call to readCode to reset orientation and code
	if (maxc > 0) {
		unit = maxu;
		ui2 = readCode(data, unit, maxa, x, y, width, height);
		c = ui2.x;
		code = ui2.y;
		code = rotateLowest(code, maxa);
	}
	return code; 
}

__global__ void cuda_krnl_topcodes(const unsigned int  *data, 
								   unsigned short      *out, 
								   int                 w, 
								   int                 h) {
	int X = (__umul24(blockIdx.x, blockDim.x) + threadIdx.x);
	int Y = (__umul24(blockIdx.y, blockDim.y) + threadIdx.y);		
	if (X>=2 && X<w-2 && Y>=2 && Y<h-2) {
		int d;
		size_t pos;
        #pragma unroll
		for (int pk=0; pk<COLS_PER_THREAD; ++pk) {
			pos = (Y*w)+(COLS_PER_THREAD*X)+pk;
			if ((data[pos]&0x2000000) && (data[pos-1]&0x2000000) && (data[pos+1]&0x2000000) &&  (data[pos-w]&0x2000000) && (data[pos+w]&0x2000000)) {
				d = cuda_krnl_decode(data, COLS_PER_THREAD*X+pk, Y, w, h);			
				if (d>=1) 
					out[pos]=(unsigned short)d;
			}		
		}
	}	
}

__host__ void gpu_scanner_compute(const unsigned int *devInRunSum, 
								  unsigned short     *devOut, 
								  int                w, 
								  int                h) {	
	const dim3 threads(BLOCKWIDTH, BLOCKHEIGHT); 
	const dim3 grid(DIVUP(w/COLS_PER_THREAD, threads.x), DIVUP(h, (threads.y)));
	cuda_krnl_topcodes <<< grid, threads >>> (devInRunSum, devOut, w, h);
}

